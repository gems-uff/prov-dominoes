#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/reduce.h>
#include <algorithm>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

//#include <cusp/format.h>
#include <cusp/complex.h>
#include <cusp/coo_matrix.h>
#include <cusp/multiply.h>
#include <cusp/print.h>

#include <hip/hip_runtime_api.h>
#include <stdio.h>

#define N_THREADS_X 8
#define N_THREADS_Y 8

__global__ void K_CheckNonZerosInCol(float *raw, int rows, int cols, int *nonZeros){
	
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (idx < rows){
		
		for (int i = 0; i < cols; i++){
			if (raw[idx * cols + i] > 0){
				nonZeros[idx] = 1;
			}
		}
	}
	
}

__global__ void K_Mean(float* mat_sum_depths, float *out_mean, int rows, int cols, float divide_by){
	
	float sum_depths  = 0;

	int columIdx = blockIdx.x * blockDim.x + threadIdx.x;

	if (columIdx < cols){
		
		
		for (int i = 0; i < rows; i++){
			sum_depths += mat_sum_depths[columIdx + (i * cols)];
		}
		out_mean[columIdx] = sum_depths / divide_by ;
	}
}

__global__ void K_Variance(float* layer, float* mean, int rows, int cols){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if ((idx < cols) && (idy < rows)){
		float meanv = mean[idx];
		float value = meanv - layer[idy * cols + idx];

		layer[idy * cols + idx] = value * value;
	}
}

__global__ void K_StandardDeviation(float* mat_sum_depths, float *out_sd, int rows, int cols, int depths){

	float sum_depths  = 0;

	int columIdx = blockIdx.x * blockDim.x + threadIdx.x;

	if (columIdx < cols){
		for (int i = 0; i < rows; i++){
			sum_depths += mat_sum_depths[columIdx + (i * cols)];
		}
		out_sd[columIdx] = sqrt(sum_depths / (rows  * depths));
	}
}

__global__ void AddKernel(float* _mat1, float *_mat2, float *_res, 
	int rows1, int cols, int cols2){
    
    int idX = blockDim.x * blockIdx.x + threadIdx.x;
    int idY = blockDim.y * blockIdx.y + threadIdx.y;
     
    
    int id = idY * cols2 + idX;
    
    if (id < rows1 * cols2){
    
    	_res[id] = 0;
    
    	int mat1_row = idY * cols;

    	for ( int i = 0; i < cols; i++){
    		int mat2_col = i * cols2 + idX;
    		_res[id] += _mat1[mat1_row + i] * _mat2[mat2_col];
    	}
    }
}

__global__ void StandardScoreKernel(float* _mat, int rows, int cols,
	float* meanSD, float *res){
    
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (idx < cols){
    
    	for (int i = 0; i < rows; i++){
    		int idxElement = i * cols + idx;    		    						
			float _mean = meanSD[idx];
			float _sd = meanSD[cols + idx];
			
			res[idxElement] = (_mat[idxElement] - _mean) / _sd;
		}
    }
}

__global__ void ConfidenceKernel(float *values, float *diagonal, int elements, float *result){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (idx < elements){
    	if (diagonal[idx] > 0)
    		result[idx] = values[idx] / diagonal[idx];
    }
}


extern "C" {

	void g_ResetAndSetGPUDevice(int gpuDevice){
		fprintf(stderr, "\nSetting GPU to %d\n", gpuDevice);
		checkCudaErrors(hipSetDevice(gpuDevice));
		checkCudaErrors(hipDeviceReset());
		checkCudaErrors(hipSetDevice(gpuDevice));
	}
	
    void g_MatMul(int n_rowsA, int n_colsA, int n_colsB, int nzA, int nzB,
    	int *rowsA, int *colsA, float *valuesA,
    	int *rowsB, int *colsB, float *valuesB,
    	int **row_res, int **col_res, float **value_res,
    	int& res_nz){
    
    	//fprintf(stderr, "before coo1\n");
    	
    	cusp::coo_matrix<int,float,cusp::host_memory> matA(n_rowsA,n_colsA,nzA);
    	for (int i = 0; i < nzA; i++){
    		matA.row_indices[i] = rowsA[i]; matA.column_indices[i] = colsA[i]; matA.values[i] = valuesA[i];
    	}
    	//fprintf(stderr, "before coo2\n");
    	cusp::coo_matrix<int,float,cusp::device_memory> matA_d = matA;
    	
    	//fprintf(stderr, "before coo3\n");
    	cusp::coo_matrix<int,float,cusp::host_memory> matB(n_colsA,n_colsB,nzB);
    	for (int i = 0; i < nzB; i++){
    		matB.row_indices[i] = rowsB[i]; matB.column_indices[i] = colsB[i]; matB.values[i] = valuesB[i];
    	}
    	//fprintf(stderr, "before coo4\n");
    	cusp::coo_matrix<int,float,cusp::device_memory> matB_d = matB;
    	
    	//fprintf(stderr, "result: r: %d, c: %d, total: %d\n", n_rowsA,n_colsB, n_rowsA * n_colsB);
    	cusp::coo_matrix<int,float,cusp::device_memory> matRes_d(n_rowsA,n_colsB, n_rowsA * n_colsB);
    	//fprintf(stderr, "before coo6\n");
    	
    	cusp::multiply(matA_d, matB_d, matRes_d);
    	//fprintf(stderr, "before coo7\n");
    	
    	cusp::coo_matrix<int,float,cusp::host_memory> matRes = matRes_d;
    	//fprintf(stderr, "before coo8\n");
    	
    	res_nz = matRes.num_entries;
    	int *_row_res = new int[res_nz];
    	int *_col_res = new int[res_nz];
    	float *_value_res = new float[res_nz];
    	
    	//fprintf(stderr, "before coo9\n");
    	
    	for(size_t n = 0; n < res_nz; n++)
  		{
    		_row_res[n] = matRes.row_indices[n];
    		_col_res[n] = matRes.column_indices[n];
   		 	_value_res[n] = matRes.values[n];
   		}
   		
   		//fprintf(stderr, "before coo10\n");

		*row_res = _row_res;
    	*col_res = _col_res;
    	*value_res = _value_res;
   		
   		//cusp::print(matRes);
    }
    
    
    void g_StandardDeviation(float* mat, int rows, int cols, 
		float* meanSD, float* result){
		
		float *d_mat;
    	float *d_meanSD;
    	float *d_result;
    	
    	checkCudaErrors(hipMalloc(&d_mat, sizeof(float) * rows * cols));
    	checkCudaErrors(hipMemcpy(d_mat, mat, sizeof(float) * rows * cols, hipMemcpyHostToDevice));
    	
    	checkCudaErrors(hipMalloc(&d_meanSD, sizeof(float) * 2 * cols));
    	checkCudaErrors(hipMemcpy(d_meanSD, meanSD, sizeof(float) * 2 * cols, hipMemcpyHostToDevice));
    	
    	checkCudaErrors(hipMalloc(&d_result, sizeof(float) * rows * cols));
    	
    	dim3 blockDim(N_THREADS_X * N_THREADS_Y, 1, 1);
        dim3 gridDim(ceil((float)cols/(N_THREADS_X *  N_THREADS_Y)), 1, 1);
        
        StandardScoreKernel<<<gridDim, blockDim>>>(d_mat, rows, cols, d_meanSD, d_result);
        
        checkCudaErrors(hipMemcpy(result, d_result, sizeof(float) * rows * cols, hipMemcpyDeviceToHost));
    	checkCudaErrors(hipFree(d_mat));
    	checkCudaErrors(hipFree(d_meanSD));
    	checkCudaErrors(hipFree(d_result));
	}
    
    
    void g_MeanSD(int rows, int cols, int depth, float *h_data, float *result, bool considerZeros){
    	// Generate the keys array
    	int h_layer_keys[rows * cols];
    	for (int i = 0; i < rows; i++){
    		
    		for (int j = 0; j < cols; j++){
    			h_layer_keys[i * cols + j] = j;
    		}
    	}


    	float *d_raw, *d_val_res, *d_mean_sd, *d_sum_depths;
    	int *d_kraw, *d_keys_res, *d_nonZeros;
    	checkCudaErrors(hipMalloc((void**) &d_raw, sizeof(float) * rows * cols));
    	checkCudaErrors(hipMalloc((void**) &d_kraw, sizeof(int) * rows * cols));
    	checkCudaErrors(hipMalloc((void**) &d_val_res, sizeof(float) * rows * cols));	
    	checkCudaErrors(hipMalloc((void**) &d_keys_res, sizeof(int) * rows * cols));
    	checkCudaErrors(hipMalloc((void**) &d_mean_sd, sizeof(float) * cols * 2));
    	checkCudaErrors(hipMalloc((void**) &d_sum_depths, sizeof(float) * depth * cols));
    	checkCudaErrors(hipMemset(d_val_res, 0, sizeof(float) * rows * cols));
    	checkCudaErrors(hipMemset(d_keys_res, 0, sizeof(int) * rows * cols));
    	
    	if (!considerZeros){
    		checkCudaErrors(hipMalloc((void**) &d_nonZeros, sizeof(int) * rows));
    		checkCudaErrors(hipMemset(d_nonZeros, 0, sizeof(int) * rows));
    	}
    		
    		


    	thrust::device_ptr<float> dev_ptr(d_raw);
    	thrust::device_ptr<int> dev_ptr_k(d_kraw);
    	thrust::device_ptr<int> dev_ptr_k_res(d_keys_res);
    	thrust::device_ptr<float> dev_ptr_v_res(d_val_res);


    	for (int i = 0; i < depth; i++){
    		checkCudaErrors(hipMemcpy(d_raw, &h_data[i * rows * cols], 
    				sizeof(float) * rows * cols, hipMemcpyHostToDevice));
    		checkCudaErrors(hipMemcpy(d_kraw, h_layer_keys, 
    				sizeof(int) * rows * cols, hipMemcpyHostToDevice));
    		
    		// check non zeros
    		if (!considerZeros){
    			dim3 blockDim_nz(256, 1, 1);
    			dim3 gridDim_nz( ceil((float)rows/256), 1, 1);
    			K_CheckNonZerosInCol<<<gridDim_nz, blockDim_nz>>>(d_raw, rows, cols, d_nonZeros);
    		}
        	       	
    		thrust::sort_by_key(dev_ptr_k, dev_ptr_k + (rows * cols), dev_ptr);
    		checkCudaErrors(hipDeviceSynchronize());
    		thrust::reduce_by_key(dev_ptr_k, dev_ptr_k+(rows * cols), dev_ptr, dev_ptr_k_res, dev_ptr_v_res);
    		checkCudaErrors(hipDeviceSynchronize());
    		checkCudaErrors(hipMemcpy(&d_sum_depths[i * cols], d_val_res, 
    				sizeof(float) * cols, hipMemcpyDeviceToDevice));
    		
    	}
    	
    	float _result[cols*depth];
    	checkCudaErrors(hipMemcpy(_result, d_mean_sd, sizeof(float) * cols * depth, hipMemcpyDeviceToHost));

    	//for (int i = 0; i < cols*depth; i++)
    	//	fprintf(stderr, "mean: %f\n", _result[i]);

    	// Calculate the mean
    	int divide_by = rows * depth;
    	
    	if (!considerZeros){
    		thrust::device_ptr<int> dev_ptr_nonZeros(d_nonZeros);
    		divide_by = thrust::reduce(dev_ptr_nonZeros, dev_ptr_nonZeros + rows) * depth;
    		//fprintf(stderr, "dividby: %d\n", divide_by);
    	}
    	checkCudaErrors(hipDeviceSynchronize());
    	
    	dim3 blockDim_m(256, 1, 1);
    	dim3 gridDim_m( ceil((float)cols/256), 1, 1);
    	K_Mean<<<gridDim_m, blockDim_m>>>(d_sum_depths, d_mean_sd, rows, cols, (float)divide_by);
    	checkCudaErrors(hipDeviceSynchronize());
    	
    	
    	/*float _result[cols];
    	checkCudaErrors(hipMemcpy(_result, d_mean_sd, sizeof(float) * cols, hipMemcpyDeviceToHost));

    	for (int i = 0; i < cols; i++)
    		fprintf(stderr, "mean: %f\n", _result[i]);*/

    	// Variance
    	/*dim3 blockDim_v(N_THREADS_X, N_THREADS_Y);
    	dim3 gridDim_v( ceil((float)cols/N_THREADS_X),ceil((float)rows/N_THREADS_Y));

    	for (int i = 0; i < depth; i++){		
    		checkCudaErrors(hipMemcpy(d_raw, &h_data[i * rows * cols], 
    				sizeof(float) * rows * cols, hipMemcpyHostToDevice));
    		checkCudaErrors(hipMemcpy(d_kraw, h_layer_keys, 
    				sizeof(int) * rows * cols, hipMemcpyHostToDevice));
        	K_Variance<<<gridDim_v, blockDim_v>>>(d_raw, d_mean_sd, rows, cols);

    		thrust::sort_by_key(dev_ptr_k, dev_ptr_k + (rows * cols), dev_ptr);
    		thrust::reduce_by_key(dev_ptr_k, dev_ptr_k+(rows * cols), dev_ptr, dev_ptr_k_res, dev_ptr_v_res);
    		checkCudaErrors(hipMemcpy(&d_sum_depths[i * cols], d_val_res, 
    				sizeof(float) * cols, hipMemcpyDeviceToDevice));
    		hipDeviceSynchronize();
    	}
    	
    	dim3 blockDim_s(256, 1, 1);
    	dim3 gridDim_s( ceil((float)cols/256), 1, 1);
    	K_StandardDeviation<<<gridDim_s, blockDim_s>>>(d_sum_depths, &d_mean_sd[cols], rows, cols, depth);
    	hipDeviceSynchronize();*/

    	
    	checkCudaErrors(hipMemcpy(result, d_mean_sd, 
    			sizeof(float) * cols * 2, hipMemcpyDeviceToHost));


    	if (!considerZeros){
    		hipFree(d_nonZeros);
    	}
    	hipFree(d_sum_depths);
    	hipFree(d_mean_sd);   	
    	hipFree(d_raw);
    	hipFree(d_kraw);
    	hipFree(d_val_res);
    	hipFree(d_keys_res);
    }
    
    
    void g_SparseMultiplication(){
    
    }
    
    void g_Confidence(float* values, float* diagonal, int elements, float* result){
		
		float *d_values;
    	float *d_diagonal;
    	float *d_result;
    	
    	checkCudaErrors(hipMalloc(&d_values, sizeof(float) * elements));
    	checkCudaErrors(hipMemcpy(d_values, values, sizeof(float) * elements, hipMemcpyHostToDevice));
    	
    	checkCudaErrors(hipMalloc(&d_diagonal, sizeof(float) * elements));
    	checkCudaErrors(hipMemcpy(d_diagonal, diagonal, sizeof(float) * elements, hipMemcpyHostToDevice));
    	
    	checkCudaErrors(hipMalloc(&d_result, sizeof(float) * elements));
    	checkCudaErrors(hipMemset(d_result, 0, sizeof(float) * elements));
    	
    	dim3 blockDim(N_THREADS_X * N_THREADS_Y, 1, 1);
        dim3 gridDim(ceil((float) elements/(N_THREADS_X *  N_THREADS_Y)), 1, 1);
        
        ConfidenceKernel<<<gridDim, blockDim>>>(d_values, d_diagonal, elements, d_result);
        
        checkCudaErrors(hipMemcpy(result, d_result, sizeof(float) * elements, hipMemcpyDeviceToHost));
    	checkCudaErrors(hipFree(d_values));
    	checkCudaErrors(hipFree(d_diagonal));
    	checkCudaErrors(hipFree(d_result));
	}
	
	bool g_IsDeviceEnabled(){		
		int nDevices;
	
		checkCudaErrors(hipGetDeviceCount(&nDevices));
	
		return nDevices > 0;
	}
    
} 