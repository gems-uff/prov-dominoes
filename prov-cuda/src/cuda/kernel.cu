#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/reduce.h>
#include <algorithm>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <cusp/complex.h>
#include <cusp/coo_matrix.h>
#include <cusp/multiply.h>
#include <cusp/print.h>

#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <math.h>

#define N_THREADS_X 16
#define N_THREADS_Y 16

__global__ void K_CheckNonZerosInCol(float *raw, int rows, int cols, int *nonZeros){
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < rows){
		for (int i = 0; i < cols; i++){
			if (raw[idx * cols + i] > 0){
				nonZeros[idx] = 1;
			}
		}
	}
}

__global__ void K_Mean(float* mat_sum_depths, float *out_mean, int rows, int cols, float divide_by){
	float sum_depths  = 0;
	int columIdx = blockIdx.x * blockDim.x + threadIdx.x;
	if (columIdx < cols){
		for (int i = 0; i < rows; i++){
			sum_depths += mat_sum_depths[columIdx + (i * cols)];
		}
		out_mean[columIdx] = sum_depths / divide_by ;
	}
}

__global__ void K_Variance(float* layer, float* mean, int rows, int cols){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if ((idx < cols) && (idy < rows)){
		float meanv = mean[idx];
		float value = meanv - layer[idy * cols + idx];
		layer[idy * cols + idx] = value * value;
	}
}

__global__ void K_StandardDeviation(float* mat_sum_depths, float *out_sd, int rows, int cols, int depths){
	float sum_depths  = 0;
	int columIdx = blockIdx.x * blockDim.x + threadIdx.x;
	if (columIdx < cols){
		for (int i = 0; i < rows; i++){
			sum_depths += mat_sum_depths[columIdx + (i * cols)];
		}
		out_sd[columIdx] = sqrt(sum_depths / (rows  * depths));
	}
}

__global__ void AddKernel(float* _mat1, float *_mat2, float *_res, 
	int rows1, int cols, int cols2){
    int idX = blockDim.x * blockIdx.x + threadIdx.x;
    int idY = blockDim.y * blockIdx.y + threadIdx.y;
    
    int id = idY * cols2 + idX;
    if (id < rows1 * cols2){
    	_res[id] = 0;
    	int mat1_row = idY * cols;
    	for ( int i = 0; i < cols; i++){
    		int mat2_col = i * cols2 + idX;
    		_res[id] += _mat1[mat1_row + i] * _mat2[mat2_col];
    	}
    }
}

__global__ void StandardScoreKernel(float* _mat, int rows, int cols,
	float* meanSD, float *res){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < cols){
    	for (int i = 0; i < rows; i++){
    		int idxElement = i * cols + idx;    		    						
			float _mean = meanSD[idx];
			float _sd = meanSD[cols + idx];
			res[idxElement] = (_mat[idxElement] - _mean) / _sd;
		}
    }
}

__global__ void ConfidenceKernel(float *values, float *diagonal, int elements, float *result){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < elements){
    	if (diagonal[idx] > 0)
    		result[idx] = values[idx] / diagonal[idx];
    }
}

// INICIO DOS PROV-KERNELS...

__global__ void binarizeKernel(float* values, int elements, float* result) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < elements) {
		if (values[idx] > 0) {
			result[idx] = 1;
		} else {
			result[idx] = 0;
		}
	}
}

__global__ void invertKernel(float* values, int elements, float* result) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < elements) {
		if (values[idx] > 0) {
			result[idx] = 0;
		} else {
			result[idx] = 1;
		}
	}
}

__global__ void diagonalizeKernel(float* values, int v, float* result) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i < v && j < v) {
		if (i==j) {
			result[i*v + j] = values[i*v + j];
		} else {
			result[i*v + j] = 0;
		}
	}
}

__global__ void upperDiagonalKernel(float* values, int v, float* result) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i < v && j < v) {
		if (j >= i) {
			result[i*v + j] = values[i*v + j];
		} else {
			result[i*v + j] = 0;
		}
	}
}

__global__ void lowerDiagonalKernel(float* values, int v, float* result) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i < v && j < v) {
		if (i >= j) {
			result[i*v + j] = values[i*v + j];
		} else {
			result[i*v + j] = 0;
		}
	}
}

__global__ void prepareClosureKernel(float* values, int v, float* result) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i < v && j < v) {
		result[i*v + j] = values[i*v + j] > 0 ? 1 : 0;
		if (i == j) {
			result[i*v + j] = 1;
		}
	}
}

__global__ void transitiveClosureKernel(float* values, int k, int v, float* result) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i < v) {
		if (j < v) {
			if (((result[i*v + k] != 0) && (result[k*v + j] != 0))) {
				if (i != j) { // ignorar próprio nó (i=j).
					float distIK = (i == k ? 0 : result[i*v + k]);
					float distKJ = (k == j ? 0 : result[k*v + j]);
					if (result[i*v + j] == 0) { // caso em que não foi calculado result entre IJ ainda.
						result[i*v + j] = distIK + distKJ;							
					} else if (distIK + distKJ < result[i*v + j]){ // atualizar se novos result forem menores que o atual.
						result[i*v + j] = distIK + distKJ;
					}
				}
			}
		}
	}
}

__global__ void rasterizeClosureKernel(float* matrix, int v) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i < v && j < v) {
		if (matrix[i*v + j] > 0) {
			matrix[i*v + j] = 1 / matrix[i*v + j];
		}
	}
}

// FIM DOS PROV-KERNELS!

extern "C" {

	void g_ResetAndSetGPUDevice(int gpuDevice) {
		checkCudaErrors(hipSetDevice(gpuDevice));
		checkCudaErrors (hipDeviceReset());checkCudaErrors
		(hipSetDevice(gpuDevice));
	}
	
	int g_getDeviceCount() {
		int nDevices = 0;
		if (hipGetDeviceCount(&nDevices) != hipSuccess) {
			hipGetLastError();
			nDevices = 0;
		}
		return nDevices;
	}
	
	bool g_IsDeviceEnabled() {
		return g_getDeviceCount()>0;
	}
	
	void g_StandardDeviation(float* mat, int rows, int cols, 
		float* meanSD, float* result){
		
		float *d_mat;
		float *d_meanSD;
		float *d_result;
		
		checkCudaErrors(hipMalloc(&d_mat, sizeof(float) * rows * cols));
		checkCudaErrors(hipMemcpy(d_mat, mat, sizeof(float) * rows * cols, hipMemcpyHostToDevice));
		
		checkCudaErrors(hipMalloc(&d_meanSD, sizeof(float) * 2 * cols));
		checkCudaErrors(hipMemcpy(d_meanSD, meanSD, sizeof(float) * 2 * cols, hipMemcpyHostToDevice));
		
		checkCudaErrors(hipMalloc(&d_result, sizeof(float) * rows * cols));
		
		dim3 blockDim(N_THREADS_X * N_THREADS_Y, 1, 1);
		dim3 gridDim(ceil((float)cols/(N_THREADS_X *  N_THREADS_Y)), 1, 1);
		
		StandardScoreKernel<<<gridDim, blockDim>>>(d_mat, rows, cols, d_meanSD, d_result);
		
		checkCudaErrors(hipMemcpy(result, d_result, sizeof(float) * rows * cols, hipMemcpyDeviceToHost));
		checkCudaErrors(hipFree(d_mat));
		checkCudaErrors(hipFree(d_meanSD));
		checkCudaErrors(hipFree(d_result));
	}
	
	
	void g_MeanSD(int rows, int cols, int depth, float *h_data, float *result, bool considerZeros){
		float *h_layer_keys;
		checkCudaErrors(hipMalloc(&h_layer_keys, sizeof(float) * rows * cols));
		
		for (int i = 0; i < rows; i++){
			
			for (int j = 0; j < cols; j++){
				h_layer_keys[i * cols + j] = j;
			}
		}

		float *d_raw, *d_val_res, *d_mean_sd, *d_sum_depths;
		int *d_kraw, *d_keys_res, *d_nonZeros;
		checkCudaErrors(hipMalloc((void**) &d_raw, sizeof(float) * rows * cols));
		checkCudaErrors(hipMalloc((void**) &d_kraw, sizeof(int) * rows * cols));
		checkCudaErrors(hipMalloc((void**) &d_val_res, sizeof(float) * rows * cols));	
		checkCudaErrors(hipMalloc((void**) &d_keys_res, sizeof(int) * rows * cols));
		checkCudaErrors(hipMalloc((void**) &d_mean_sd, sizeof(float) * cols * 2));
		checkCudaErrors(hipMalloc((void**) &d_sum_depths, sizeof(float) * depth * cols));
		checkCudaErrors(hipMemset(d_val_res, 0, sizeof(float) * rows * cols));
		checkCudaErrors(hipMemset(d_keys_res, 0, sizeof(int) * rows * cols));
		
		if (!considerZeros){
			checkCudaErrors(hipMalloc((void**) &d_nonZeros, sizeof(int) * rows));
			checkCudaErrors(hipMemset(d_nonZeros, 0, sizeof(int) * rows));
		}

		thrust::device_ptr<float> dev_ptr(d_raw);
		thrust::device_ptr<int> dev_ptr_k(d_kraw);
		thrust::device_ptr<int> dev_ptr_k_res(d_keys_res);
		thrust::device_ptr<float> dev_ptr_v_res(d_val_res);

		for (int i = 0; i < depth; i++){
			checkCudaErrors(hipMemcpy(d_raw, &h_data[i * rows * cols], 
					sizeof(float) * rows * cols, hipMemcpyHostToDevice));
			checkCudaErrors(hipMemcpy(d_kraw, h_layer_keys, 
					sizeof(int) * rows * cols, hipMemcpyHostToDevice));
			
			if (!considerZeros){
				dim3 blockDim_nz(256, 1, 1);
				dim3 gridDim_nz( ceil((float)rows/256), 1, 1);
				K_CheckNonZerosInCol<<<gridDim_nz, blockDim_nz>>>(d_raw, rows, cols, d_nonZeros);
			}
					
			thrust::sort_by_key(dev_ptr_k, dev_ptr_k + (rows * cols), dev_ptr);
			checkCudaErrors(hipDeviceSynchronize());
			thrust::reduce_by_key(dev_ptr_k, dev_ptr_k+(rows * cols), dev_ptr, dev_ptr_k_res, dev_ptr_v_res);
			checkCudaErrors(hipDeviceSynchronize());
			checkCudaErrors(hipMemcpy(&d_sum_depths[i * cols], d_val_res, sizeof(float) * cols, hipMemcpyDeviceToDevice));
		}
		
		float *_result;
		checkCudaErrors(hipMalloc(&_result, sizeof(float) * cols * depth));
		checkCudaErrors(hipMemcpy(_result, d_mean_sd, sizeof(float) * cols * depth, hipMemcpyDeviceToHost));

		// Calculate the mean
		int divide_by = rows * depth;
		
		if (!considerZeros){
			thrust::device_ptr<int> dev_ptr_nonZeros(d_nonZeros);
			divide_by = thrust::reduce(dev_ptr_nonZeros, dev_ptr_nonZeros + rows) * depth;
		}
		checkCudaErrors(hipDeviceSynchronize());
		
		dim3 blockDim_m(256, 1, 1);
		dim3 gridDim_m( ceil((float)cols/256), 1, 1);
		K_Mean<<<gridDim_m, blockDim_m>>>(d_sum_depths, d_mean_sd, rows, cols, (float)divide_by);
		checkCudaErrors(hipDeviceSynchronize());
		
		checkCudaErrors(hipMemcpy(result, d_mean_sd, sizeof(float) * cols * 2, hipMemcpyDeviceToHost));

		if (!considerZeros){
			hipFree(d_nonZeros);
		}
		hipFree(d_sum_depths);
		hipFree(d_mean_sd);   	
		hipFree(d_raw);
		hipFree(d_kraw);
		hipFree(d_val_res);
		hipFree(d_keys_res);
		hipFree(_result);
		hipFree(h_layer_keys);
	}
	
	void g_Confidence(float* values, float* diagonal, int elements, float* result){
		float *d_values;
		float *d_diagonal;
		float *d_result;
		
		checkCudaErrors(hipMalloc(&d_values, sizeof(float) * elements));
		checkCudaErrors(hipMemcpy(d_values, values, sizeof(float) * elements, hipMemcpyHostToDevice));
		
		checkCudaErrors(hipMalloc(&d_diagonal, sizeof(float) * elements));
		checkCudaErrors(hipMemcpy(d_diagonal, diagonal, sizeof(float) * elements, hipMemcpyHostToDevice));
		
		checkCudaErrors(hipMalloc(&d_result, sizeof(float) * elements));
		checkCudaErrors(hipMemset(d_result, 0, sizeof(float) * elements));
		
		dim3 blockDim(N_THREADS_X * N_THREADS_Y, 1, 1);
		dim3 gridDim(ceil((float) elements/(N_THREADS_X *  N_THREADS_Y)), 1, 1);
		
		ConfidenceKernel<<<gridDim, blockDim>>>(d_values, d_diagonal, elements, d_result);
		
		checkCudaErrors(hipMemcpy(result, d_result, sizeof(float) * elements, hipMemcpyDeviceToHost));
		checkCudaErrors(hipFree(d_values));
		checkCudaErrors(hipFree(d_diagonal));
		checkCudaErrors(hipFree(d_result));
	}
		
	
	void g_Binarize(float* values, int elements, float* result) {
		float* d_values;
		float* d_result;
		checkCudaErrors(hipMalloc(&d_values, sizeof(float) * elements));
		checkCudaErrors(
				hipMemcpy(d_values, values, sizeof(float) * elements,
						hipMemcpyHostToDevice));
	
		checkCudaErrors(hipMalloc(&d_result, sizeof(float) * elements));
		checkCudaErrors(hipMemset(d_result, 0, sizeof(float) * elements));
	
		dim3 blockDim(N_THREADS_X * N_THREADS_Y, 1);
		dim3 gridDim(ceil((float) elements / (N_THREADS_X * N_THREADS_X)), 1, 1);
	
		binarizeKernel<<<gridDim, blockDim>>>(d_values, elements, d_result);
	
		checkCudaErrors(
				hipMemcpy(result, d_result, sizeof(float) * elements,
						hipMemcpyDeviceToHost));
	
		checkCudaErrors(hipFree(d_values));
		checkCudaErrors(hipFree(d_result));
	}
	
	void g_Invert(float* values, int elements, float* result) {
		float* d_values;
		float* d_result;
		checkCudaErrors(hipMalloc(&d_values, sizeof(float) * elements));
		checkCudaErrors(
				hipMemcpy(d_values, values, sizeof(float) * elements,
						hipMemcpyHostToDevice));
	
		checkCudaErrors(hipMalloc(&d_result, sizeof(float) * elements));
		checkCudaErrors(hipMemset(d_result, 0, sizeof(float) * elements));
	
		dim3 blockDim(N_THREADS_X * N_THREADS_Y, 1);
		dim3 gridDim(ceil((float) elements / (N_THREADS_X * N_THREADS_X)), 1, 1);
	
		invertKernel<<<gridDim, blockDim>>>(d_values, elements, d_result);
	
		checkCudaErrors(
				hipMemcpy(result, d_result, sizeof(float) * elements,
						hipMemcpyDeviceToHost));
	
		checkCudaErrors(hipFree(d_values));
		checkCudaErrors(hipFree(d_result));
	}
	
	void g_Diagonalize(float* values, int v, float* result) {
		float* d_values;
		float* d_result;
		checkCudaErrors(hipMalloc(&d_values, sizeof(float) * v*v));
		checkCudaErrors(
				hipMemcpy(d_values, values, sizeof(float) * v*v,
						hipMemcpyHostToDevice));
	
		checkCudaErrors(hipMalloc(&d_result, sizeof(float) * v*v));
		checkCudaErrors(hipMemset(d_result, 0, sizeof(float) * v*v));
	
		dim3 blockDim(N_THREADS_X , N_THREADS_Y, 1);
		dim3 gridDim(ceil((float) v / (N_THREADS_X)), ceil((float) v / (N_THREADS_Y)), 1);
	
		diagonalizeKernel<<<gridDim, blockDim>>>(d_values, v, d_result);
	
		checkCudaErrors(
				hipMemcpy(result, d_result, sizeof(float) * v*v,
						hipMemcpyDeviceToHost));
	
		checkCudaErrors(hipFree(d_values));
		checkCudaErrors(hipFree(d_result));
	}
	
	void g_UpperDiagonal(float* values, int v, float* result) {
		float* d_values;
		float* d_result;
		checkCudaErrors(hipMalloc(&d_values, sizeof(float) * v*v));
		checkCudaErrors(
				hipMemcpy(d_values, values, sizeof(float) * v*v,
						hipMemcpyHostToDevice));
	
		checkCudaErrors(hipMalloc(&d_result, sizeof(float) * v*v));
		checkCudaErrors(hipMemset(d_result, 0, sizeof(float) * v*v));
	
		dim3 blockDim(N_THREADS_X , N_THREADS_Y, 1);
		dim3 gridDim(ceil((float) v / (N_THREADS_X)), ceil((float) v / (N_THREADS_Y)), 1);
	
		upperDiagonalKernel<<<gridDim, blockDim>>>(d_values, v, d_result);
	
		checkCudaErrors(
				hipMemcpy(result, d_result, sizeof(float) * v*v,
						hipMemcpyDeviceToHost));
	
		checkCudaErrors(hipFree(d_values));
		checkCudaErrors(hipFree(d_result));
		}
	
	void g_LowerDiagonal(float* values, int v, float* result) {
			float* d_values;
			float* d_result;
			checkCudaErrors(hipMalloc(&d_values, sizeof(float) * v*v));
			checkCudaErrors(
					hipMemcpy(d_values, values, sizeof(float) * v*v,
							hipMemcpyHostToDevice));
		
			checkCudaErrors(hipMalloc(&d_result, sizeof(float) * v*v));
			checkCudaErrors(hipMemset(d_result, 0, sizeof(float) * v*v));
		
			dim3 blockDim(N_THREADS_X , N_THREADS_Y, 1);
			dim3 gridDim(ceil((float) v / (N_THREADS_X)), ceil((float) v / (N_THREADS_Y)), 1);
		
			lowerDiagonalKernel<<<gridDim, blockDim>>>(d_values, v, d_result);
		
			checkCudaErrors(
					hipMemcpy(result, d_result, sizeof(float) * v*v,
							hipMemcpyDeviceToHost));
		
			checkCudaErrors(hipFree(d_values));
			checkCudaErrors(hipFree(d_result));
		}
	
	void g_TransitiveClosure(float* values, int v, float* result) {
		float* d_values;
		float* d_result;
		checkCudaErrors(hipMalloc(&d_values, sizeof(float) * v*v));
		checkCudaErrors(
				hipMemcpy(d_values, values, sizeof(float) * v*v,
						hipMemcpyHostToDevice));
	
		checkCudaErrors(hipMalloc(&d_result, sizeof(float) * v*v));
		checkCudaErrors(hipMemset(d_result, 0, sizeof(float) * v*v));
	
		
		dim3 blockDim(N_THREADS_X, N_THREADS_Y, 1);
		dim3 gridDim(ceil((float) v / (N_THREADS_X)), ceil((float) v / (N_THREADS_Y)), 1);
		
		prepareClosureKernel<<<gridDim, blockDim>>>(d_values, v, d_result);
		
		for (int k=0; k < v; k++) {
			transitiveClosureKernel<<<gridDim, blockDim>>>(d_values, k, v, d_result);
		}
		
		rasterizeClosureKernel<<<gridDim, blockDim>>>(d_result, v);
	
		checkCudaErrors(
				hipMemcpy(result, d_result, sizeof(float) * v*v,
						hipMemcpyDeviceToHost));
	
		checkCudaErrors(hipFree(d_values));
		checkCudaErrors(hipFree(d_result));
	}
	
	void g_MatMul(int n_rowsA, int n_colsA, int n_colsB, int nzA, int nzB,
		int *rowsA, int *colsA, float *valuesA,
		int *rowsB, int *colsB, float *valuesB,
		int **row_res, int **col_res, float **value_res,
		int& res_nz){

		
		cusp::coo_matrix<int,float,cusp::host_memory> matA(n_rowsA,n_colsA,nzA);
		for (int i = 0; i < nzA; i++){
			matA.row_indices[i] = rowsA[i]; matA.column_indices[i] = colsA[i]; matA.values[i] = valuesA[i];
		}
		cusp::coo_matrix<int,float,cusp::device_memory> matA_d = matA;
		
		cusp::coo_matrix<int,float,cusp::host_memory> matB(n_colsA,n_colsB,nzB);
		for (int i = 0; i < nzB; i++){
			matB.row_indices[i] = rowsB[i]; matB.column_indices[i] = colsB[i]; matB.values[i] = valuesB[i];
		}
		cusp::coo_matrix<int,float,cusp::device_memory> matB_d = matB;
		
		cusp::coo_matrix<int,float,cusp::device_memory> matRes_d(n_rowsA,n_colsB, n_rowsA * n_colsB);
		
		cusp::multiply(matA_d, matB_d, matRes_d);
		
		cusp::coo_matrix<int,float,cusp::host_memory> matRes = matRes_d;
		
		res_nz = matRes.num_entries;
		int *_row_res = new int[res_nz];
		int *_col_res = new int[res_nz];
		float *_value_res = new float[res_nz];
		
		for(size_t n = 0; n < res_nz; n++)
		{
			_row_res[n] = matRes.row_indices[n];
			_col_res[n] = matRes.column_indices[n];
			_value_res[n] = matRes.values[n];
		}
		
		*row_res = _row_res;
		*col_res = _col_res;
		*value_res = _value_res;
	}
}
